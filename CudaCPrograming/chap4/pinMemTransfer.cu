#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char**argv)
{
	int dev = 0;
	hipSetDevice(dev);

	unsigned int isize = 1<<22;
	unsigned int nbytes = isize * sizeof(float);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("device %dL %s memory size %d nbytes %5.2fMB\n", dev, deviceProp.name, isize, nbytes/(1024.0f * 1024.0f));

	float *h_a = (float*) malloc(nbytes);
	hipHostMalloc((float**)&h_a, nbytes);
	for (unsigned int i = 0; i < nbytes; i++)
		h_a[i] = 0.5f;

	float *d_a;
	hipMalloc((float**)&d_a, nbytes);

	hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(h_a, d_a, nbytes, cudaMemcpyDevideToHost);

	hipFree(d_a);
	hipHostFree(h_a);

	hipDeviceReset();
	return EXIT_SUCCESS;
}
