#include "hip/hip_runtime.h"
#include "../common//common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NSTREAM 4
#define BDIM 128

void initialData(float *ip, int size)
{
	int i;
	for(int i = 0; i < size; i++)
		ip[i] = (float) std::rand() & 0xFF / 10.0f;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
	for(int idx = 0; idx < N; idx++)
		C[idx] = A[idx] + B[idx];
}

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		for(int i = 0; i < N; i++)
			C[idx] = A[idx] + B[idx];
	}
}

void checkResults(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	for(int i = 0; i < N; i++)
	{
		if (abs(hostRef[i] - gpuRef) > epsilon)
		{
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}
	if (match) printf("Arrays match.\n\n");
}

int main(int argc, char **argv)
{
	printf("> %s Starting...\n", argv[0]);

	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("> Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	if(deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
	{
		if(deviceProp.concurrentKernels == 0)
		{
			printf("> GPU does not support concurrent kernel execution (SM3.5 or higher required)\n");
			printf("> CUDA kernel runs will be serialized\n");
		}
		else
		{
			printf("> GPU does not support Hyper Q\n");
			printf("> GPU kernel runs will have limited concurrency\n");
		}
	}
	printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);


	char *iname = "CUDA_DEVICE_MAX_CONNECTIONS";
	setenv(iname, "1", 1);
	char *ivalue = getenv(iname);
	printf("> %s=%s\n", iname, ivalue);
	printf("> with streams=%d\n", NSTREAM);

	int nElem = 1 << 18;
	printf("> vector size = %d\n", nElem);
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
	CHECK(hipHostAlloc((void**)&h_B, nBytes, hipHostMallocDefault));
	CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));
	CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));

	initialData(h_A, nElem);
	initialData(h_B, nElem);
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	float *d_A, *d_B, *d_C;
	CHECK(hipMalloc((float **)&d_A, nBytes));
	CHECK(hipMalloc((float **)&d_B, nBytes));
	CHECK(hipMalloc((float **)&d_C, nBytes));

	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	dim3 block(BDIM);
	dim3 grid((nElem + block.x - 1) / block.x);
	printf("> grid (%d, %d) block (%d, %d)\n", grid.x, grid.y, block.x, block.y);

	CHECK(hipEventRecord(start, 0));
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	CHECK(hipEventRecord(stop, 0));
	CHECK(cudaEventSynchonize(stop));
	float memcpy_h2d_time;
	CHECK(hipEventElapsedTime(&memcpy_h2d_time, start, stop));

	CHECK(hipEventRecord(start, 0));
	sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);
	CHECK(hipEventRecord(stop, 0));
	CHECK(cudaEventSynchonize(stop));
	float kernel_time;
	CHECK(hipEventElapsedTime(&kernel_time, start, stop));

	CHECK(hipEventRecord(start, 0));
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipEventRecord(stop, 0));
	CHECK(cudaEventSynchonize(stop));
	float memcpy_d2h_time;
	CHECK(hipEventElapsedTime(&memcpy_d2h_time, start, stop));
	float itotal = memcpy_h2d_time + kernel_time + memcpy_d2h_time;

	printf("\n");
	printf("Measured timings (throughput): \n");
	printf("Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time, (nBytes * 1e-6) / memcpy_h2d_time);
	printf("Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time, (nBytes * 1e-6) / memcpy_d2h_time);
	printf("Kernel\t\t\t: %f ms (%f GB/s)\n", kernel_time, (nBytes * 2e-6) / kernel_time);
	printf("Total\t\t\t: %f ms (%f GB/s)\n", itotal, (nBytes * 2e-6) / itotal);

	// grid parallel 
	int iElem = nElem / NSTREAM;
	size_t iBytes = iElem * sizeof(float);
	grid.x = (iElem + block.x - 1) / block.x;

	hipStream_t stream[NSTREAM];
	for(int i = 0; i < NSTREAM; i++)
		CHECK(hipStreamCreate(&stream[i]));

	CHECK(hipEventRecord(start, 0));
	for(int i = 0; i < NSTREAM; i++)
	{
		int ioffset = i * iElem;
		CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
		CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
	}

	for(int i = 0; i < NSTREAM; i++)
	{
		int ioffset = i * iElem;
		sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset], *d_C[ioffset], iElem);
	}


	for(int i = 0; i < NSTREAM; i++)
	{
		int ioffset = i * iElem;
		CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]));
	}

	CHECK(hipEventRecord(stop, 0));
	CHECK(cudaEventSynchonize(stop));
	float execution_time;
	CHECK(hipEventElapsedTime(&execution_time, start, stop));

	printf("\n");
	printf("Actual results from overlapped data transfers:\n");
	printf(" overlap with %d streams: %f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6)/execution_time);
	printf("Speedup			:%f\n", ((itotal -execution_time) * 100.0f) / itotal);

	CHECK(hipGetLastError());

	checkResults(hostRef, gpuRef, nElem);

	//free memory
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));
	CHECK(hipHostFree(h_A));
	CHECK(hipHostFree(h_B));
	CHECK(hipHostFree(hostRef));
	CHECK(hipHostFree(gpuRef));

	CHECK(hipEventDestroy(start));
	CHECK(hipEventDestroy(stop));

	for(int i = 0; i < NSTREAM; i++)
		CHECK(hipStreamDestroy(streams[i]));

	CHECK(hipDeviceReset());

	return 0;
}
